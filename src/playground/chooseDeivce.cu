
#include <hip/hip_runtime.h>
#include <iostream>

int main(void)
{
  hipDeviceProp_t prop;
  int dev;
  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 7;
  prop.minor = 5;
  hipChooseDevice(&dev, &prop);
  printf("ID of CUDA device closest to revision %d.%d: %d\n", prop.major, prop.minor, dev);
}